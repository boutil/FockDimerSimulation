
#include <hip/hip_runtime.h>
extern "C"
__global__ void flipFaces(int *faceStates, float *faceWeights, int *insideBoundary, float *rand, int size)
{
    int i = (blockIdx.x * blockDim.x + threadIdx.x) / size;
    int j = (blockIdx.x * blockDim.x + threadIdx.x) % size;
    int index = i*size + j;


    if((index < 3*size*size) && (insideBoundary[index] == 1)) {
        float prop = 0;
        if (faceStates[index] == 42) {
            prop = faceWeights[index];
        } else if (faceStates[index] == 21) {
            prop = 1/faceWeights[index];
        }
        if ( rand[index] < prop * 0.9999 ) {
            if ( faceStates[index] == 21 ) { 
                faceStates[index] = 42;
            }
            else if ( faceStates[index] == 42 ) { 
                faceStates[index] = 21;
            }
        }
    }
}

// otherFaceStates 1 and 2 are those of parity one and two larger than theseFaceStates.
// We assume a 3Nx3N grid.
// We use (f(parity) % 3) / 2 to produce a number that's 1 for only one of the parities.
extern "C"
__global__ void consolidateFaces(int *theseFaceStates, int *otherFaceStates1, int *otherFaceStates2, int size, int parity)
{
    int i = ((blockIdx.x * blockDim.x + threadIdx.x) / size) + 1;
    int j = ((blockIdx.x * blockDim.x + threadIdx.x) % size) + 1;
    int index = i*size + j;
    
    if(i < 3 * size - 1 && j < size - 1) {
        theseFaceStates[index] = (otherFaceStates1[(i-1)*size+j] & 1) * 8                                    // W
                                    + (otherFaceStates2[(i+1)*size+j] & 8) / 8                               // E
                                    + (otherFaceStates2[i*size+j - ((parity + i + 2) % 3) / 2] & 2) * 8      // SW
                                    + (otherFaceStates1[i*size+j + ((parity + i) % 3) / 2] & 16) / 8         // NE
                                    + (otherFaceStates1[(i+1)*size+j - ((parity + i + 2) % 3) / 2] & 4) * 8  // SE
                                    + (otherFaceStates2[(i-1)*size+j + ((parity + i) % 3) / 2] & 32) / 8;    // NW
    }
}