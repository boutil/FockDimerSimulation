
#include <hip/hip_runtime.h>

extern "C"
__global__ void flipFaces(int *faceStates, float *faceWeights, int *insideBoundary, float *rand, int size)
{
    int i = (blockIdx.x * blockDim.x + threadIdx.x) / size;
    int j = (blockIdx.x * blockDim.x + threadIdx.x) % size;
    int index = i*size + j;
    // faceStates[index] = index;


    if((index < 2*size*size) && (insideBoundary[index] == 1)) {
        float prop = 0;
        if (faceStates[index] == 10) {
            prop = faceWeights[index];
        } else if (faceStates[index] == 5) {
            prop = 1/faceWeights[index];
        }
        if ( rand[index] < prop * 0.9999 ) {
            if ( faceStates[index] == 5 ) { 
                faceStates[index] = 10;
            }
            else if ( faceStates[index] == 10 ) { 
                faceStates[index] = 5;
            }
        }
    }
}


extern "C"
__global__ void consolidateFaces(int *theseFaceStates, int *otherFaceStates, int size, int parity)
{
    int i = ((blockIdx.x * blockDim.x + threadIdx.x) / size) + 1;
    int j = ((blockIdx.x * blockDim.x + threadIdx.x) % size) + 1;
    int index = i*size + j;
    
    if(i < 2 * size - 1 && j < size - 1) {
        theseFaceStates[index] = (otherFaceStates[(i-1)*size+j] & 4)/4
                                    + 4*(otherFaceStates[(i+1)*size+j] & 1) 
                                    + (otherFaceStates[i*size+j - (parity + i + 1) % 2] & 8)/4 
                                    + 4*(otherFaceStates[i*size+j + (parity + i) % 2] & 2);
    }
}